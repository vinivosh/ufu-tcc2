
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__
void add(long n, float *x, float *y) {
  int index = threadIdx.x;
  int stride = blockDim.x;

  for (long i = index; i < n; i += stride)
    y[i] += x[i];
}

int main(void){
  long N = long(1<<28) + long(1<<27); // 402.653.184 elementos

  float *x, *y;
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  for (long i = 0; i < N; i++) {
    x[i] = 3.77f; y[i] = 3.23f;
  }

  add<<<1, 1024>>>(N, x, y);
  hipDeviceSynchronize();

  float maxError = 0.0f;
  for (long i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i] - 7.0f));
  std::cout << "Max error: " << maxError << "\n";

  hipFree(x);
  hipFree(y);
  
  return 0;
}