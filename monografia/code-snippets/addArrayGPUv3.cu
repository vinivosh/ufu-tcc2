
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__
void add(int n, float *x, float *y){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] += x[i];
}

int main(void){
  int N = 1<<28; // 268.435.456 elementos

  float *x, *y;
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 3.77f; y[i] = 3.23f;
  }

  int blockSize = 1024;
  int numBlocks = ceil((N + blockSize - 1) / blockSize);

  add<<<numBlocks, blockSize>>>(N, x, y);
  hipDeviceSynchronize();

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i] - 7.0f));
  std::cout << "Max error: " << maxError << "\n";

  hipFree(x);
  hipFree(y);
  
  return 0;
}